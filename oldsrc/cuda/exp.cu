#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>
#include "../r.h"
#include "../exp.h"

int
runexp(int n) {
	int i;
	size_t s;
	uint *a, *b, *c;

	s = n*sizeof(uint);
	/* note we're not checking for possible mem errors */
	hipHostAlloc((void **)&a, s, hipHostMallocWriteCombined);
	hipHostAlloc((void **)&b, s, hipHostMallocWriteCombined);
	hipHostAlloc((void **)&c, s, hipHostMallocPortable);
	for(i = 0; i < n; i++) {
		a[i] = rand() % UINT_MAX/2;
		b[i] = rand() % UINT_MAX/2;
		c[i] = 0;
	}
	randsum(n, a, b, c);

	/* output validation */
	for(i = 0; i < n; i++) {
		if(c[i] != a[i] + b[i]) {
			fprintf(stderr, "invalid output @ %d (a: %d, b: %d, c: %d)", i, a[i], b[i], c[i]);
			return n;
		}
	}

	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
	return 0;
}
