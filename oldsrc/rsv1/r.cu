#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void
k_randsum(int n, uint *a, uint *b, uint *c) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < n) {
		c[i] = a[i] + b[i];
	}
}

__host__ void
hostmalloc(void **ptr, size_t size, uint flags) {
	hipHostAlloc(ptr, size, flags);
}

__host__ void
hostfree(void **ptr) {
	cudaHostFree(ptr);
}


