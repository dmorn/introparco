#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "event.h"

void
start(Event *e, hipStream_t s) {
	hipEventRecord(e->tic, s);
}

float
stop(Event *e, hipStream_t s) {
	float elaps;
	hipEventRecord(e->toc, s);
	hipEventSynchronize(e->toc);
	hipEventElapsedTime(&elaps, e->tic, e->toc);

	return elaps;
}

void
freeevent(Event *e) {
	hipEventDestroy(e->tic);
	hipEventDestroy(e->toc);
	free(e);
}

Event*
newevent(void) {
	Event *e;
	e = (Event*) malloc(sizeof(Event));
	hipEventCreate(&(e->tic));
	hipEventCreate(&(e->toc));
	return e;
}
