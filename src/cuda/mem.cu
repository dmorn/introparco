#include <stdlib.h>
#include <stddef.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../dat.h"
#include "../fns.h"

void
allocsum(int n, uint **a, uint **b, uint **c) {
	size_t s = sizeof(uint)*n;
	/* notice we're not checking for errors */
	hipHostAlloc(a, s, hipHostMallocWriteCombined);
	hipHostAlloc(b, s, hipHostMallocWriteCombined);
	hipHostAlloc(c, s, hipHostMallocDefault);
}

void
hfree(void *ptr) {
	hipHostFree(ptr);
}

