#include <stdlib.h>
#include <stddef.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../dat.h"
#include "../fns.h"

void
sumalloc(int n, uint **a, uint **b, uint **c) {
	size_t s = sizeof(uint)*n;
	/* notice we're not checking for errors */
	hipHostAlloc(a, s, hipHostMallocWriteCombined);
	hipHostAlloc(b, s, hipHostMallocWriteCombined);
	hipHostAlloc(c, s, hipHostMallocDefault);
}

void
hostfree(void *ptr) {
	hipHostFree(ptr);
}

