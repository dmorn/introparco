#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../r.h"
#include "../exp.h"

__global__ void
k_randsum(int n, uint *a, uint *b, uint *c) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < n) {
		c[i] = a[i] + b[i];
	}
}

void
randsum(int n, uint *a, uint *b, uint *c) {
	uint flags;
	uint *da, *db, *dc;
	size_t s;
	int thd, blk;

	flags = hipHostRegisterMapped;
	s = n*sizeof(uint);
	hipHostRegister(a, s, flags);
	hipHostRegister(b, s, flags);
	hipHostRegister(c, s, flags);
	hipMalloc(&da, s);
	hipMalloc(&db, s);
	hipMalloc(&dc, s);

	hipMemcpy(da, a, s, hipMemcpyHostToDevice);
	hipMemcpy(db, b, s, hipMemcpyHostToDevice);

	thd = 256;
	blk = (n+thd-1)/thd;

	fprintf(stderr, "blk: %d, thd: %d\n", blk, thd);
	k_randsum<<<blk, thd>>>(n, da, db, dc);
	hipMemcpy(c, dc, s, hipMemcpyDeviceToHost);

	hipHostUnregister(a);
	hipHostUnregister(b);
	hipHostUnregister(c);
	hipFree(da);
	hipFree(db);
	hipFree(dc);
}

