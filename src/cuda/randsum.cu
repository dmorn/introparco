#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../randsum.h"
#include "../exp.h"

char *expdesc = "randsum cuda with cuda kernel";

__global__
void
k_randsum(int n, uint *a, uint *b, uint *c) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < n) {
		c[i] = a[i] + b[i];
	}
}

void
randsum(int n, uint *a, uint *b, uint *c) {
	uint *da, *db, *dc;
	size_t s;
	int thd, blk;

	s = n*sizeof(uint);
	hipMalloc(&da, s);
	hipMalloc(&db, s);
	hipMalloc(&dc, s);

	hipMemcpy(da, a, s, hipMemcpyHostToDevice);
	hipMemcpy(db, b, s, hipMemcpyHostToDevice);

	thd = 256;
	blk = (n+thd-1)/thd;

	if(debug)
		fprintf(stderr, "thd: %d, blk: %d\n", thd, blk);
	k_randsum<<<thd, blk>>>(n, da, db, dc);
	hipMemcpy(c, dc, s, hipMemcpyDeviceToHost);

	hipFree(da);
	hipFree(db);
	hipFree(dc);
}

