#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../dat.h"
#include "../fns.h"

__global__ void
k_sum(int n, uint *a, uint *b, uint *c) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < n) {
		c[i] = a[i] + b[i];
	}
}

void
sum(Msr *lp, int n, uint a[], uint b[], uint c[]) {
	uint *da, *db, *dc;
	size_t s;
	int thd, blk;
	double tic;
	Msr m = {MuNS, "sum", 0, NULL};
	Msr m1 = {MuNS, "cudaMemcpyHtD", 0, NULL};
	Msr m2 = {MuNS, "hipMalloc", 0, NULL};
	Msr m3 = {MuNS, "cudaMallocDtH", 0, NULL};

	tic = now();
	s = n*sizeof(uint);
	hipMalloc(&da, s);
	hipMalloc(&db, s);
	hipMalloc(&dc, s);
	m2.val = (uint)(now()-tic);
	addmsr(lp, &m2);

	tic = now();
	hipMemcpy(da, a, s, hipMemcpyHostToDevice);
	hipMemcpy(db, b, s, hipMemcpyHostToDevice);
	m1.val = (uint)(now()-tic);
	addmsr(lp, &m1);

	thd = 32;
	blk = (n+thd-1)/thd;

	tic = now();
	k_sum<<<blk, thd>>>(n, da, db, dc);
	m.val = (uint)(now()-tic);
	addmsr(lp, &m);

	tic = now();
	hipMemcpy(c, dc, s, hipMemcpyDeviceToHost);
	m3.val = (uint)(now()-tic);
	addmsr(lp, &m3);

	hipFree(da);
	hipFree(db);
	hipFree(dc);
}

