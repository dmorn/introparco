#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "event.h"
#include "../dat.h"
#include "../fns.h"

__global__ void
k_sum(int n, uint *a, uint *b, uint *c) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < n) {
		c[i] = a[i] + b[i];
	}
}

void
sum(Msr *lp, int n, uint a[], uint b[], uint c[]) {
	size_t s;
	uint *da, *db, *dc;
	int thd, blk;
	Event *e;

	e = newevent();
	s = n*sizeof(uint);
	hipMalloc(&da, s);
	hipMalloc(&db, s);
	hipMalloc(&dc, s);

	start(e, 0);
	hipMemcpy(da, a, s, hipMemcpyHostToDevice);
	hipMemcpy(db, b, s, hipMemcpyHostToDevice);
	addmsr(lp, newmsr(UnitMS, "cudaMemcpyHtD", stop(e, 0)));

	thd = 32;
	blk = (n+thd-1)/thd;

	start(e, 0);
	k_sum<<<blk, thd>>>(n, da, db, dc);
	addmsr(lp, newmsr(UnitMS, "k_sum", stop(e, 0)));

	start(e, 0);
	hipMemcpy(c, dc, s, hipMemcpyDeviceToHost);
	addmsr(lp, newmsr(UnitMS, "cudaMemcpyDtH", stop(e, 0)));

	hipFree(da);
	hipFree(db);
	hipFree(dc);
	freeevent(e);
}
