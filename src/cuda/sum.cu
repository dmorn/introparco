#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../dat.h"
#include "../fns.h"

__global__ void
k_sum(int n, uint *a, uint *b, uint *c) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < n) {
		c[i] = a[i] + b[i];
	}
}

void
sum(Msr *lp, int n, uint a[], uint b[], uint c[]) {
	size_t s;
	uint *da, *db, *dc;
	int thd, blk;
	hipEvent_t start, stop;
	float elaps;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	s = n*sizeof(uint);
	hipMalloc(&da, s);
	hipMalloc(&db, s);
	hipMalloc(&dc, s);

	hipEventRecord(start, 0);
	hipMemcpy(da, a, s, hipMemcpyHostToDevice);
	hipMemcpy(db, b, s, hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elaps, start, stop);
	addmsr(lp, newmsr(UnitMS, "cudaMemcpyHtD", elaps));

	thd = 32;
	blk = (n+thd-1)/thd;

	hipEventRecord(start, 0);
	k_sum<<<blk, thd>>>(n, da, db, dc);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elaps, start, stop);
	addmsr(lp, newmsr(UnitMS, "k_sum", elaps));

	hipEventRecord(start, 0);
	hipMemcpy(c, dc, s, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elaps, start, stop);
	addmsr(lp, newmsr(UnitMS, "cudaMemcpyDtH", elaps));

	hipFree(da);
	hipFree(db);
	hipFree(dc);
}
