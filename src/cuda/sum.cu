#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../dat.h"
#include "../fns.h"

__global__ void
k_sum(int n, uint *a, uint *b, uint *c) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < n) {
		c[i] = a[i] + b[i];
	}
}

void
sum(Msr *lp, int n, uint a[], uint b[], uint c[]) {
	uint *da, *db, *dc;
	size_t s;
	int thd, blk;
	double tic;
	Msr *m1, *m2, *m3, *m4;
	Msr m = {MuNS, "sum", 0, NULL};
	Msr m1 = {MuNS, "cudaMemcpyHtD", 0, NULL};
	Msr m2 = {MuNS, "hipMalloc", 0, NULL};
	Msr m3 = {MuNS, "cudaMallocDtH", 0, NULL};

	tic = now();
	s = n*sizeof(uint);
	hipMalloc(&da, s);
	hipMalloc(&db, s);
	hipMalloc(&dc, s);
	m1 = msrnew(MuNS, "hipMalloc", (uint)(now()-tic));
	msradd(lp, m1);

	tic = now();
	hipMemcpy(da, a, s, hipMemcpyHostToDevice);
	hipMemcpy(db, b, s, hipMemcpyHostToDevice);
	m2 = msrnew(MuNS, "cudaMemcpyHtoD", (uint)(now()-tic));
	msradd(lp, m2);

	thd = 32;
	blk = (n+thd-1)/thd;

	tic = now();
	k_sum<<<blk, thd>>>(n, da, db, dc);
	m3 = msrnew(MuNS, "sum", (uint)(now()-tic));
	msradd(lp, m3);

	tic = now();
	hipMemcpy(c, dc, s, hipMemcpyDeviceToHost);
	m4 = msrnew(MuNS, "cudaMemcpyDtoH", (uint)(now()-tic));
	msradd(lp, m4);

	hipFree(da);
	hipFree(db);
	hipFree(dc);
}

