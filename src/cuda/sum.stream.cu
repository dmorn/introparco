#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../dat.h"
#include "../fns.h"

void
sum(Msr *lp, int n, uint a[], uint b[], uint c[]) {
	uint *da, *db, *dc;
	size_t s;

	s = n*sizeof(uint);
	hipMalloc(&da, s);
	hipMalloc(&db, s);
	hipMalloc(&dc, s);
}
