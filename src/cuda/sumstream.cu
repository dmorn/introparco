#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "event.h"
#include "../dat.h"
#include "../fns.h"

const int maxstreamsize = 1e6

__global__ void
k_sum(int n, uint *a, uint *b, uint *c) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < n) {
		c[i] = a[i] + b[i];
	}
}

void
sum(Msr *lp, int n, uint a[], uint b[], uint c[]) {
	size_t size;
	uint *da, *db, *dc, i, offset;
	int thd, blk, nstreams, nbytes, streamsize;
	Event *e;
	hipStream_t *streams;

	e = newevent();

	size = n*sizeof(uint);
	streamsize = (maxstreamsize < n) ? maxstreamsize : n;
	nstreams = n / streamsize;
	nbytes = n / nstreams;
	streams = malloc(sizeof(hipStream_t)*nstreams);
	for(i = 0; i < nstreams; i++) {
		hipStreamCreate(streams[i]);
	}

	thd = 32;
	blk = (n+thd-1)/thd;

	hipMalloc(&da, size);
	hipMalloc(&db, size);
	hipMalloc(&dc, size);

	for(i = 0; i < nstreams; i++) {
		offset = i * streamsize;
		start(e, *streams[i]);
		hipMemcpyAsync(da[offset], a[offset], streamsize, hipMemcpyHostToDevice, stream);
		addmsr(lp, newmsr(UnitMS, "cudaMemcpyHtD", stop(e, *stream[i])));

		start(e, *stream[i]);
		k_sum<<<blk, thd, 0, *streams[i]>>>(streamsize, da[offset], db[offset], dc[offset]);
		addmsr(lp, newmsr(UnitMS, "k_sum", stop(e, *streams[i])));

		start(e, *stream[i]);
		hipMemcpyAsync(c[offset], dc[offset], streamsize, hipMemcpyDeviceToHost, *stream[i]);
		addmsr(lp, newmsr(UnitMS, "cudaMemcpyDtH", stop(e, *streams[i])));
	}

	for(i = 0; i < nstreams; i++) {
		hipStreamSynchronize(*streams[i]);
		hipStreamDestroy(*streams[i]);
	}

	hipFree(da);
	hipFree(db);
	hipFree(dc);
	hipStreamDestroy(stream);
	freeevent(e);
	free(streams);
}
