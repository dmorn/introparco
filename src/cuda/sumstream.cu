#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "event.h"
#include "../dat.h"
#include "../fns.h"

const int maxstreamsize = 1e6;

__global__ void
k_sum(int n, int offset, uint *a, uint *b, uint *c) {
	int i = offset + blockDim.x * blockIdx.x + threadIdx.x;
	if(i < n) {
		c[i] = a[i] + b[i];
	}
}

void
sum(Msr *lp, int n, uint a[], uint b[], uint c[]) {
	size_t size;
	uint *da, *db, *dc, i, todo, doing;
	int thd, blk, nstreams, offset;
	Event *e;
	hipStream_t *streams;
	uint *streamsize;

	e = newevent();

	size = n*sizeof(uint);
	hipMalloc(&da, size);
	hipMalloc(&db, size);
	hipMalloc(&dc, size);

	nstreams = (n+maxstreamsize-1)/maxstreamsize;
	streams = (hipStream_t*) malloc(sizeof(hipStream_t)*nstreams);
	streamsize = (uint*) malloc(sizeof(uint)*nstreams);

	todo = n;
	for(i = 0; i < nstreams; i++) {
		hipStreamCreate(&streams[i]);
		if(todo < maxstreamsize)
			doing = todo;
		else
			doing = maxstreamsize;

		todo -= doing;
		streamsize[i] = doing;
	}

	thd = 32;
	blk = (n+thd-1)/thd;

	fprintf(stderr, "running with %d streams\n", nstreams);
	for(i = 0, offset = 0; i < nstreams; offset += streamsize[i], i++) {
		size = streamsize[i]*sizeof(uint);
		fprintf(stderr, "i: %d, size: %u, offset: %u\n", i, size, offset);

		start(e, streams[i]);
		hipMemcpyAsync(&da[offset], &a[offset], size, hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(&db[offset], &b[offset], size, hipMemcpyHostToDevice, streams[i]);
		addmsr(lp, newmsr(UnitMS, "cudaMemcpyHtD", stop(e, streams[i])));

		start(e, streams[i]);
		k_sum<<<blk, thd, 0, streams[i]>>>(n, offset, da, db, dc);
		addmsr(lp, newmsr(UnitMS, "k_sum", stop(e, streams[i])));

		start(e, streams[i]);
		hipMemcpyAsync(&c[offset], &dc[offset], size, hipMemcpyDeviceToHost, streams[i]);
		addmsr(lp, newmsr(UnitMS, "cudaMemcpyDtH", stop(e, streams[i])));
	}

	for(i = 0; i < nstreams; i++) {
		hipStreamSynchronize(streams[i]);
		hipStreamDestroy(streams[i]);
	}

	hipFree(da);
	hipFree(db);
	hipFree(dc);
	freeevent(e);
	free(streams);
	free(streamsize);
}
