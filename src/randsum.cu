#include "hip/hip_runtime.h"
#include <stdint.h>
#include <limits.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "randsum.h"

__global__
void k_randsum(int n, uint32_t *a, uint32_t *b, uint32_t *c) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < n) {
		c[i] = a[i] + b[i];
	}
}

extern "C" void randsum(int n, uint32_t *a, uint32_t *b, uint32_t *c) {
	uint32_t *da, *db, *dc;
	size_t s;
	int thd, blk;

	for (int i = 0; i < n; i++) {
		a[i] = rand() % UINT32_MAX/2;
		b[i] = rand() % UINT32_MAX/2;
	}

	s = n*sizeof(uint32_t);
	hipMalloc(&da, s);
	hipMalloc(&db, s);
	hipMalloc(&dc, s);

	hipMemcpy(da, a, s, hipMemcpyHostToDevice);
	hipMemcpy(db, b, s, hipMemcpyHostToDevice);

	thd = 256;
	blk = (n+thd-1)/thd;

	k_randsum<<<thd, blk>>>(n, da, db, dc);
	hipMemcpy(c, dc, s, hipMemcpyDeviceToHost);

	hipFree(da);
	hipFree(db);
	hipFree(dc);
}
